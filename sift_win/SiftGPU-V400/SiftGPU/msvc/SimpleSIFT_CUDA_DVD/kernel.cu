#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}


#include <stdlib.h>
#include <vector>
#include <iostream>
using std::vector;
using std::iostream;


////////////////////////////////////////////////////////////////////////////
#if !defined(SIFTGPU_STATIC) && !defined(SIFTGPU_DLL_RUNTIME) 
// SIFTGPU_STATIC comes from compiler
#define SIFTGPU_DLL_RUNTIME
// Load at runtime if the above macro defined
// comment the macro above to use static linking
#endif

////////////////////////////////////////////////////////////////////////////
// define REMOTE_SIFTGPU to run computation in multi-process (Or remote) mode
// in order to run on a remote machine, you need to start the server manually
// This mode allows you use Multi-GPUs by creating multiple servers
// #define REMOTE_SIFTGPU
// #define REMOTE_SERVER        NULL
// #define REMOTE_SERVER_PORT   7777


///////////////////////////////////////////////////////////////////////////
//#define DEBUG_SIFTGPU  //define this to use the debug version in windows

#ifdef _WIN32
#ifdef SIFTGPU_DLL_RUNTIME
#define WIN32_LEAN_AND_MEAN
#include <windows.h>
#define FREE_MYLIB FreeLibrary
#define GET_MYPROC GetProcAddress
#else
//define this to get dll import definition for win32
#define SIFTGPU_DLL
#ifdef _DEBUG 
#pragma comment(lib, "../../lib/siftgpu_d.lib")
#else
#pragma comment(lib, "../../lib/siftgpu.lib")
#endif
#endif
#else
#ifdef SIFTGPU_DLL_RUNTIME
#include <dlfcn.h>
#define FREE_MYLIB dlclose
#define GET_MYPROC dlsym
#endif
#endif

#include "../SiftGPU/SiftGPU.h"


int main()
{
#ifdef SIFTGPU_DLL_RUNTIME
#ifdef _WIN32
#ifdef _DEBUG
	HMODULE  hsiftgpu = LoadLibrary("siftgpu_d.dll");
#else
	HMODULE  hsiftgpu = LoadLibrary("siftgpu.dll");
#endif
#else
	void * hsiftgpu = dlopen("libsiftgpu.so", RTLD_LAZY);
#endif

	if (hsiftgpu == NULL) return 0;

#ifdef REMOTE_SIFTGPU
	ComboSiftGPU* (*pCreateRemoteSiftGPU) (int, char*) = NULL;
	pCreateRemoteSiftGPU = (ComboSiftGPU* (*) (int, char*)) GET_MYPROC(hsiftgpu, "CreateRemoteSiftGPU");
	ComboSiftGPU * combo = pCreateRemoteSiftGPU(REMOTE_SERVER_PORT, REMOTE_SERVER);
	SiftGPU* sift = combo;
	SiftMatchGPU* matcher = combo;
#else
	SiftGPU* (*pCreateNewSiftGPU)(int) = NULL;
	SiftMatchGPU* (*pCreateNewSiftMatchGPU)(int) = NULL;
	pCreateNewSiftGPU = (SiftGPU* (*) (int)) GET_MYPROC(hsiftgpu, "CreateNewSiftGPU");
	pCreateNewSiftMatchGPU = (SiftMatchGPU* (*)(int)) GET_MYPROC(hsiftgpu, "CreateNewSiftMatchGPU");
	SiftGPU* sift = pCreateNewSiftGPU(1);
	SiftMatchGPU* matcher = pCreateNewSiftMatchGPU(4096);
#endif

#elif defined(REMOTE_SIFTGPU)
	ComboSiftGPU * combo = CreateRemoteSiftGPU(REMOTE_SERVER_PORT, REMOTE_SERVER);
	SiftGPU* sift = combo;
	SiftMatchGPU* matcher = combo;
#else
	//this will use overloaded new operators
	SiftGPU  *sift = new SiftGPU;
	SiftMatchGPU *matcher = new SiftMatchGPU(4096);
#endif
	vector<float > descriptors1(1), descriptors2(1);
	vector<SiftGPU::SiftKeypoint> keys1(1), keys2(1);
	int num1 = 0, num2 = 0;

	//process parameters
	//The following parameters are default in V340
	//-m,       up to 2 orientations for each feature (change to single orientation by using -m 1)
	//-s        enable subpixel subscale (disable by using -s 0)


	char * argv[] = { "-fo", "-1", "-v", "1" };//
	//-fo -1    staring from -1 octave 
	//-v 1      only print out # feature and overall time
	//-loweo    add a (.5, .5) offset
	//-tc <num> set a soft limit to number of detected features

	//NEW:  parameters for  GPU-selection
	//1. CUDA.                   Use parameter "-cuda", "[device_id]"
	//2. OpenGL.				 Use "-Display", "display_name" to select monitor/GPU (XLIB/GLUT)
	//   		                 on windows the display name would be something like \\.\DISPLAY4

	//////////////////////////////////////////////////////////////////////////////////////
	//You use CUDA for nVidia graphic cards by specifying
	//-cuda   : cuda implementation (fastest for smaller images)
	//          CUDA-implementation allows you to create multiple instances for multiple threads
	//          Checkout src\TestWin\MultiThreadSIFT
	/////////////////////////////////////////////////////////////////////////////////////

	//////////////////////////////////////////////////////////////////////////////////////
	////////////////////////Two Important Parameters///////////////////////////
	// First, texture reallocation happens when image size increases, and too many 
	// reallocation may lead to allocatoin failure.  You should be careful when using 
	// siftgpu on a set of images with VARYING imag sizes. It is recommended that you 
	// preset the allocation size to the largest width and largest height by using function
	// AllocationPyramid or prameter '-p' (e.g. "-p", "1024x768").

	// Second, there is a parameter you may not be aware of: the allowed maximum working
	// dimension. All the SIFT octaves that needs a larger texture size will be skipped.
	// The default prameter is 2560 for the unpacked implementation and 3200 for the packed.
	// Those two default parameter is tuned to for 768MB of graphic memory. You should adjust
	// it for your own GPU memory. You can also use this to keep/skip the small featuers.
	// To change this, call function SetMaxDimension or use parameter "-maxd".
	//
	// NEW: by default SiftGPU will try to fit the cap of GPU memory, and reduce the working 
	// dimension so as to not allocate too much. This feature can be disabled by -nomc
	//////////////////////////////////////////////////////////////////////////////////////


	int argc = sizeof(argv) / sizeof(char*);
	sift->ParseParam(argc, argv);

	///////////////////////////////////////////////////////////////////////
	//Only the following parameters can be changed after initialization (by calling ParseParam). 
	//-dw, -ofix, -ofix-not, -fo, -unn, -maxd, -b
	//to change other parameters at runtime, you need to first unload the dynamically loaded libaray
	//reload the libarary, then create a new siftgpu instance


	//Create a context for computation, and SiftGPU will be initialized automatically 
	//The same context can be used by SiftMatchGPU
	if (sift->CreateContextGL() != SiftGPU::SIFTGPU_FULL_SUPPORTED) return 0;

	if (sift->RunSIFT("../data/800-1.jpg"))
	{
		//Call SaveSIFT to save result to file, the format is the same as Lowe's
		//sift->SaveSIFT("../data/800-1.sift"); //Note that saving ASCII format is slow

		//get feature count
		num1 = sift->GetFeatureNum();

		//allocate memory
		keys1.resize(num1);    descriptors1.resize(128 * num1);

		//reading back feature vectors is faster than writing files
		//if you dont need keys or descriptors, just put NULLs here
		sift->GetFeatureVector(&keys1[0], &descriptors1[0]);
		//this can be used to write your own sift file.            
	}

	//You can have at most one OpenGL-based SiftGPU (per process).
	//Normally, you should just create one, and reuse on all images. 
	if (sift->RunSIFT("../data/640-1.jpg"))
	{
		num2 = sift->GetFeatureNum();
		keys2.resize(num2);    descriptors2.resize(128 * num2);
		sift->GetFeatureVector(&keys2[0], &descriptors2[0]);
	}

	//Testing code to check how it works when image size varies
	//sift->RunSIFT("../data/256.jpg");sift->SaveSIFT("../data/256.sift.1");
	//sift->RunSIFT("../data/1024.jpg"); //this will result in pyramid reallocation
	//sift->RunSIFT("../data/256.jpg"); sift->SaveSIFT("../data/256.sift.2");
	//two sets of features for 256.jpg may have different order due to implementation

	//*************************************************************************
	/////compute descriptors for user-specified keypoints (with or without orientations)

	//Method1, set new keypoints for the image you've just processed with siftgpu
	//say vector<SiftGPU::SiftKeypoint> mykeys;
	//sift->RunSIFT(mykeys.size(), &mykeys[0]); 
	//sift->RunSIFT(num2, &keys2[0], 1);         sift->SaveSIFT("../data/640-1.sift.2");
	//sift->RunSIFT(num2, &keys2[0], 0);        sift->SaveSIFT("../data/640-1.sift.3");

	//Method2, set keypoints for the next coming image
	//The difference of with method 1 is that method 1 skips gaussian filtering
	//SiftGPU::SiftKeypoint mykeys[100];
	//for(int i = 0; i < 100; ++i){
	//    mykeys[i].s = 1.0f;mykeys[i].o = 0.0f;
	//    mykeys[i].x = (i%10)*10.0f+50.0f;
	//    mykeys[i].y = (i/10)*10.0f+50.0f;
	//}
	//sift->SetKeypointList(100, mykeys, 0);
	//sift->RunSIFT("../data/800-1.jpg");                    sift->SaveSIFT("../data/800-1.sift.2");
	//### for comparing with method1: 
	//sift->RunSIFT("../data/800-1.jpg"); 
	//sift->RunSIFT(100, mykeys, 0);                          sift->SaveSIFT("../data/800-1.sift.3");
	//*********************************************************************************


	//**********************GPU SIFT MATCHING*********************************
	//**************************select shader language*************************
	//SiftMatchGPU will use the same shader lanaguage as SiftGPU by default
	//Before initialization, you can choose between glsl, and CUDA(if compiled). 
	//matcher->SetLanguage(SiftMatchGPU::SIFTMATCH_CUDA); // +i for the (i+1)-th device

	//Verify current OpenGL Context and initialize the Matcher;
	//If you don't have an OpenGL Context, call matcher->CreateContextGL instead;
	matcher->VerifyContextGL(); //must call once

	//Set descriptors to match, the first argument must be either 0 or 1
	//if you want to use more than 4096 or less than 4096
	//call matcher->SetMaxSift() to change the limit before calling setdescriptor
	matcher->SetDescriptors(0, num1, &descriptors1[0]); //image 1
	matcher->SetDescriptors(1, num2, &descriptors2[0]); //image 2

	//match and get result.    
	int(*match_buf)[2] = new int[num1][2];
	//use the default thresholds. Check the declaration in SiftGPU.h
	int num_match = matcher->GetSiftMatch(num1, match_buf);
	std::cout << num_match << " sift matches were found;\n";

	//enumerate all the feature matches
	for (int i = 0; i < num_match; ++i)
	{
		//How to get the feature matches: 
		//SiftGPU::SiftKeypoint & key1 = keys1[match_buf[i][0]];
		//SiftGPU::SiftKeypoint & key2 = keys2[match_buf[i][1]];
		//key1 in the first image matches with key2 in the second image
	}

	//*****************GPU Guided SIFT MATCHING***************
	//example: define a homography, and use default threshold 32 to search in a 64x64 window
	//float h[3][3] = {{0.8f, 0, 0}, {0, 0.8f, 0}, {0, 0, 1.0f}};
	//matcher->SetFeatureLocation(0, &keys1[0]); //SetFeatureLocaiton after SetDescriptors
	//matcher->SetFeatureLocation(1, &keys2[0]);
	//num_match = matcher->GetGuidedSiftMatch(num1, match_buf, h, NULL);
	//std::cout << num_match << " guided sift matches were found;\n";
	//if you can want to use a Fundamental matrix, check the function definition

	// clean up..
	delete[] match_buf;
#ifdef REMOTE_SIFTGPU
	delete combo;
#else
	delete sift;
	delete matcher;
#endif

#ifdef SIFTGPU_DLL_RUNTIME
	FREE_MYLIB(hsiftgpu);
#endif
	return 1;
}

/*
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
*/